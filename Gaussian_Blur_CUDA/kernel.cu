#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>

using std::cout;
using std::fstream;

int dummy; // makes it work

#define BLOCK_SIZE_MULTIPLIER 4 // TODO: make ths dynamic as to not exceed 1024 blocks
#define BLOCK_SIZE_LINEAR 2000

#define CLAMP(x, a, b) ((x) < (a) ? (a) : ((x) > (b) ? (b) : (x)))
#define DIVCEIL(x, y) (((x) + (y) - 1) / (y)) // division that rounds up

__global__ void separateChannels(uchar4 *inputRGBA,
                                 int *_x_dim,
                                 int *_y_dim,
                                 unsigned char *r,
                                 unsigned char *g,
                                 unsigned char *b)
{
    int x_dim = *_x_dim;
    int y_dim = *_y_dim;

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= x_dim * y_dim) {
        return;  // value out of bounds, don't do anything
    }
    uchar4 tmp = inputRGBA[i];
    r[i] = (unsigned char)tmp.x;
    g[i] = (unsigned char)tmp.y;
    b[i] = (unsigned char)tmp.z;
}

__global__ void recombineChannels(unsigned char *r,
                                  unsigned char *g,
                                  unsigned char *b,
                                  int *_x_dim,
                                  int *_y_dim,
                                  uchar4 *outputRGBA)
{
    int x_dim = *_x_dim;
    int y_dim = *_y_dim;

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= x_dim * y_dim) {
        return;
    }
    uchar4 tmp;
    tmp.x = r[i];
    tmp.y = g[i];
    tmp.z = b[i];

    tmp.w = 255;  // no transparency
    outputRGBA[i] = tmp;
}

__global__ void gaussianBlur(unsigned char *in,
                             float *filter,
                             int *_filter_dim,
                             int *_x_dim,
                             int *_y_dim,
                             unsigned char *out)
{
    int filter_dim = *_filter_dim;
    int x_dim = *_x_dim;
    int y_dim = *_y_dim;

    int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int y_pos = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_pos >= x_dim || y_pos >= y_dim) {
        return;  // out of bounds
    }

    float val = 0.0f;

    int x_i_pos, y_i_pos, offset = filter_dim / 2;
    for (int x_i = 0; x_i < filter_dim; x_i++) {
        for (int y_i = 0; y_i < filter_dim; y_i++) {
            // multiply each value in the adjacent pixels clamped to the edges
            // of the image by the corresponding filter value and add it to the
            // total value that will be set in the blurred image
            x_i_pos = x_i + x_pos - offset;
            y_i_pos = y_i + y_pos - offset;
            val += float(in[CLAMP(x_i_pos, 0, x_dim - 1) +
                            CLAMP(y_i_pos, 0, y_dim - 1) * x_dim]) *
                filter[x_i + y_i * filter_dim];
        }
    }
    out[x_pos + y_pos * x_dim] = (unsigned char)val;
}

int main()
{
    // initialize, allocate and read relevant host image variables
    int h_x_dim, h_y_dim;
    uchar4 *h_img;

    fstream h_img_stream_in;
    h_img_stream_in.open("test.ppm", fstream::in);
    h_img_stream_in.ignore(2, EOF); // ignores the P3 at the beginning of the file
    h_img_stream_in >> h_x_dim >> h_y_dim;
    h_img_stream_in >> dummy;
    
    int h_xy_dim = h_x_dim * h_y_dim;
    h_img = (uchar4 *) malloc(h_xy_dim * sizeof(uchar4));
    
    int x, y, z;
    for (int i = 0; i < h_xy_dim; i++) {
        h_img_stream_in >> x >> y >> z;
        h_img[i].x = (unsigned char)x;
        h_img[i].y = (unsigned char)y;
        h_img[i].z = (unsigned char)z;
        h_img[i].w = 255;
    }

    h_img_stream_in.close();

    // initialize, allocate and read relevant host filter variables
    int h_filter_dim;
    float *h_filter;
    fstream h_filter_stream_in;
    h_filter_stream_in.open("filter", fstream::in);
    h_filter_stream_in >> h_filter_dim;

    int h_filter_dim2 = h_filter_dim * h_filter_dim;
    h_filter = (float *) malloc(h_filter_dim2 * sizeof(float));

    for (int i = 0; i < h_filter_dim2; i++) {
        h_filter_stream_in >> h_filter[i];
    }

    h_filter_stream_in.close();

    // initialie and allocate relevant device variables
    uchar4 *d_img_in, *d_img_out;
    unsigned char *d_img_r_in, *d_img_g_in, *d_img_b_in;
    unsigned char *d_img_r_out, *d_img_g_out, *d_img_b_out;
    int *d_x_dim, *d_y_dim;

    float *d_filter;
    int *d_filter_dim;

    hipMalloc((void **) &d_img_in, h_xy_dim * sizeof(uchar4));
    hipMalloc((void **) &d_img_out, h_xy_dim * sizeof(uchar4));
    hipMalloc((void **) &d_img_r_in, h_xy_dim * sizeof(unsigned char));
    hipMalloc((void **) &d_img_g_in, h_xy_dim * sizeof(unsigned char));
    hipMalloc((void **) &d_img_b_in, h_xy_dim * sizeof(unsigned char));
    hipMalloc((void **) &d_img_r_out, h_xy_dim * sizeof(unsigned char));
    hipMalloc((void **) &d_img_g_out, h_xy_dim * sizeof(unsigned char));
    hipMalloc((void **) &d_img_b_out, h_xy_dim * sizeof(unsigned char));
    hipMalloc((void **) &d_x_dim, sizeof(int));
    hipMalloc((void **) &d_y_dim, sizeof(int));
    hipMalloc((void **) &d_filter, h_filter_dim2 * sizeof(float));
    hipMalloc((void **) &d_filter_dim, sizeof(int));

    hipMemcpy(d_img_in, h_img, h_xy_dim * sizeof(uchar4), hipMemcpyHostToDevice);
    hipMemcpy(d_x_dim, &h_x_dim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y_dim, &h_y_dim, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, h_filter_dim2 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter_dim, &h_filter_dim, sizeof(int), hipMemcpyHostToDevice);

    // determine appropriate block dimensions and numbers
    dim3 block_dim = dim3(h_filter_dim * BLOCK_SIZE_MULTIPLIER,
                          h_filter_dim * BLOCK_SIZE_MULTIPLIER, 1);
    dim3 block_number = dim3(DIVCEIL(h_x_dim, block_dim.x),
                             DIVCEIL(h_y_dim, block_dim.y), 1);

    cout<<"starting operations\n";
    // perform operations on GPU

    separateChannels<<<BLOCK_SIZE_LINEAR, DIVCEIL(h_xy_dim, BLOCK_SIZE_LINEAR)>>>
        (d_img_in, d_x_dim, d_y_dim, d_img_r_in, d_img_g_in, d_img_b_in);
    cout<<"channels separated\n";
    
    gaussianBlur<<<block_dim, block_number>>>
        (d_img_r_in, d_filter, d_filter_dim, d_x_dim, d_y_dim, d_img_r_out);
    cout<<"red blurred\n";
    gaussianBlur<<<block_dim, block_number>>>
        (d_img_g_in, d_filter, d_filter_dim, d_x_dim, d_y_dim, d_img_g_out);
    cout<<"green blurred\n";
    gaussianBlur<<<block_dim, block_number>>>
        (d_img_b_in, d_filter, d_filter_dim, d_x_dim, d_y_dim, d_img_b_out);
    cout<<"blue blurred\n";
    
    recombineChannels<<<BLOCK_SIZE_LINEAR, DIVCEIL(h_xy_dim, BLOCK_SIZE_LINEAR)>>>
        (d_img_r_out, d_img_g_out, d_img_b_out, d_x_dim, d_y_dim, d_img_out);
    cout<<"channels recombined\n";
    cout<<"operations done.\n";

    // copy data back from GPU and print it to file
    hipMemcpy(h_img, d_img_out, h_xy_dim * sizeof(uchar4), hipMemcpyDeviceToHost);

    fstream h_img_stream_out;
    h_img_stream_out.open("blurred.ppm", fstream::out);
    h_img_stream_out << "P3 " << h_x_dim << " " << h_y_dim << "\n255\n";
    for (int i = 0; i < h_xy_dim; i++) {
        h_img_stream_out << (int)h_img[i].x << " " << (int)h_img[i].y << " " << (int)h_img[i].z << "\n";
    }
    h_img_stream_out.close();

    // free memory and exit
    free(h_img);
    free(h_filter);

    hipFree(d_img_in);
    hipFree(d_img_out);
    hipFree(d_img_r_in);
    hipFree(d_img_g_in);
    hipFree(d_img_b_in);
    hipFree(d_img_r_out);
    hipFree(d_img_g_out);
    hipFree(d_img_b_out);
    hipFree(d_x_dim);
    hipFree(d_y_dim);
    hipFree(d_filter);
    hipFree(d_filter_dim);

    return 0;
}